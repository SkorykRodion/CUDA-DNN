
#include "matrix.cuh"


Matrix::Matrix(size_t x_dim, size_t y_dim) :
        shape(x_dim, y_dim), data_device(nullptr), data_host(nullptr),
        device_allocated(false), host_allocated(false)
{ }

Matrix::Matrix(Shape shape) :
        Matrix(shape.x, shape.y)
{ }

void Matrix::allocateCudaMemory() {
    if (!device_allocated) {
        float* device_memory = nullptr;
        hipMalloc(&device_memory, shape.x * shape.y * sizeof(float));
        NNException::throwIfDeviceErrorsOccurred("Cannot allocate CUDA memory for Tensor3D.");
        data_device = std::shared_ptr<float>(device_memory,
                                             [&](float* ptr){ hipFree(ptr); });
        device_allocated = true;
    }
}

void Matrix::allocateHostMemory() {
    if (!host_allocated) {
        data_host = std::shared_ptr<float>(new float[shape.x * shape.y],
                                           [&](float* ptr){ delete[] ptr; });
        host_allocated = true;
    }
}

void Matrix::allocateMemory() {
    allocateCudaMemory();
    allocateHostMemory();
}

void Matrix::allocateMemoryIfNotAllocated(Shape shape) {
    if (!device_allocated && !host_allocated) {
        this->shape = shape;
        allocateMemory();
    }
}

void Matrix::allocateMemoryIfNotAllocatedZero(Shape shape) {
    if (!device_allocated && !host_allocated) {
        this->shape = shape;
        allocateMemory();
        setZeroHost();
        setZeroDevice();
    }
}

void Matrix::copyHostToDevice() {
    if (device_allocated && host_allocated) {
        hipMemcpy(data_device.get(), data_host.get(), shape.x * shape.y * sizeof(float), hipMemcpyHostToDevice);
        NNException::throwIfDeviceErrorsOccurred("Cannot copy host data to CUDA device.");
    }
    else {
        throw NNException("Cannot copy host data to not allocated memory on device.");
    }
}

void Matrix::copyDeviceToHost() {
    if (device_allocated && host_allocated) {
        hipMemcpy(data_host.get(), data_device.get(), shape.x * shape.y * sizeof(float), hipMemcpyDeviceToHost);
        NNException::throwIfDeviceErrorsOccurred("Cannot copy device data to host.");
    }
    else {
        throw NNException("Cannot copy device data to not allocated memory on host.");
    }
}

float& Matrix::operator[](const int index) {
    return data_host.get()[index];
}

const float& Matrix::operator[](const int index) const {
    return data_host.get()[index];
}

void Matrix::print() const {
    std::cout << "Host data:" << std::endl;
    for (size_t row = 0; row < shape.y; ++row) {
        for (size_t col = 0; col < shape.x; ++col) {
            std::cout << std::setprecision(4) << data_host.get()[row * shape.x + col] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Device data:" << std::endl;
    float* host_data = new float[shape.x * shape.y];
    hipMemcpy(host_data, data_device.get(), shape.x * shape.y * sizeof(float), hipMemcpyDeviceToHost);
    for (size_t row = 0; row < shape.y; ++row) {
        for (size_t col = 0; col < shape.x; ++col) {
            std::cout << std::setprecision(4) << host_data[row * shape.x + col] << " ";
        }
        std::cout << std::endl;
    }
    delete[] host_data;
}

void Matrix::setZeroHost() {
    if (!host_allocated) {
        allocateHostMemory();
    }
    std::fill(data_host.get(), data_host.get() + shape.x*shape.y, 0.0f);
}

void Matrix::setZeroDevice() {
    if (!device_allocated) {
        allocateCudaMemory();
    }
    hipMemset(data_device.get(), 0, shape.x*shape.y * sizeof(float));
}


Matrix::Matrix(const Matrix& other) : shape(other.shape) {
    device_allocated = false;
    if (other.device_allocated) {
        allocateCudaMemory();
        hipMemcpy(data_device.get(), other.data_device.get(), other.shape.x*other.shape.y * sizeof(float), hipMemcpyDeviceToDevice);
        device_allocated = true;
    }
    host_allocated = false;
    if (other.host_allocated) {
        allocateHostMemory();
        std::copy(other.data_host.get(), other.data_host.get() + other.shape.x*other.shape.y, data_host.get());
        host_allocated = true;
    }
}

bool Matrix::isDeviceAllocated() const {
    return device_allocated;
}

bool Matrix::isHostAllocated() const {
    return host_allocated;
}
